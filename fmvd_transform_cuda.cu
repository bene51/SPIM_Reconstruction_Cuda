#include "hip/hip_runtime.h"
#include <windows.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "fmvd_utils.h"
#include "fmvd_transform_cuda.h"

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <hipfft/hipfft.h>

#define checkCudaErrors(ans) {__gpuAssert((ans), __FILE__, __LINE__); }

void
__gpuAssert(unsigned int code, const char *file, int line, bool abort=true)
{
	if(code != hipSuccess) {
		const char *str = hipGetErrorString((hipError_t)code);
		fprintf(stderr, "GPUAssert: error %d %s %d\n", code, file, line);
		fprintf(stderr, "%s\n", str);
		if(abort)
			exit(code);
	}
}

texture<unsigned short, 3, hipReadModeNormalizedFloat> tex;

__global__ void
transform_data_kernel(
		unsigned short *dTransformed,
		int z,
		int w,
		int h,
		int d,
		int wTransformed,
		int hTransformed,
		float *inv_matrix)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if(x < wTransformed && y < hTransformed) {
		float rx = inv_matrix[0] * x + inv_matrix[1] * y + inv_matrix[2]  * z + inv_matrix[3];
		float ry = inv_matrix[4] * x + inv_matrix[5] * y + inv_matrix[6]  * z + inv_matrix[7];
		float rz = inv_matrix[8] * x + inv_matrix[9] * y + inv_matrix[10] * z + inv_matrix[11];

		/*
		// mirror
		if(rx < 0) rx = -rx; if(rx > w - 1) rx = 2 * w - rx - 2;
		if(ry < 0) ry = -ry; if(ry > h - 1) ry = 2 * h - ry - 2;
		if(rz < 0) rz = -rz; if(rz > d - 1) rz = 2 * d - rz - 2;
		*/

		float v = tex3D(tex, rx + 0.5, ry + 0.5, rz + 0.5);
		unsigned short iv = (unsigned short)(v * 65535 + 0.5);
		dTransformed[y * wTransformed + x] = iv;
	}
}

__global__ void
transform_mask_kernel(
		unsigned short *dTransformed,
		int z,
		int w,
		int h,
		int d,
		float zspacing,
		int wTransformed,
		int hTransformed,
		float border,
		float *inv_matrix)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if(x < wTransformed && y < hTransformed) {
		float rx = inv_matrix[0] * x + inv_matrix[1] * y + inv_matrix[2]  * z + inv_matrix[3];
		float ry = inv_matrix[4] * x + inv_matrix[5] * y + inv_matrix[6]  * z + inv_matrix[7];
		float rz = inv_matrix[8] * x + inv_matrix[9] * y + inv_matrix[10] * z + inv_matrix[11];

		if(rx < 0 || rx >= w || ry < 0 || ry >= h || rz < 0 || rz >= d) {
			dTransformed[y * wTransformed + x] = 0;
		} else {
			float v = 1;
			float dx = rx < w / 2 ? rx : w - rx;
			float dy = ry < h / 2 ? ry : h - ry;
			float dz = rz < d / 2 ? rz : d - rz;
			dz *= zspacing;

			if(dx < border)
				v = v * (0.5f * (1 - cos(dx / border * HIP_PI_F)));
			if(dy < border)
				v = v * (0.5f * (1 - cos(dy / border * HIP_PI_F)));
			if(dz < border)
				v = v * (0.5f * (1 - cos(dz / border * HIP_PI_F)));
			dTransformed[y * wTransformed + x] = (unsigned short)(65535 * v + 0.5);
		}
	}
}


void transform_cuda(
		unsigned short *h_data,
		int w,
		int h,
		int d,
		int tw,
		int th,
		int td,
		float *h_inverse,
		const char *outfile,
		int createTransformedMasks,
		int border,
		float zspacing,
		const char *maskfile)
{
	size_t data_size = w * h * d * sizeof(unsigned short);
	hipArray *d_data = 0;
	unsigned short *d_transformed;
	unsigned short *h_transformed;
	float *d_inverse;

	int plane_size = tw * th * sizeof(unsigned short);

	const hipExtent volumeSize = make_hipExtent(w, h, d);
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned short>();
	checkCudaErrors(hipMalloc3DArray(&d_data, &desc, volumeSize));
	checkCudaErrors(hipHostMalloc((void **)&h_transformed, plane_size));

	printf("td = %d\n", td);


	// copy data to 3D array
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void *)h_data, w * sizeof(unsigned short), w, h);
	copyParams.dstArray = d_data;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	tex.normalized = false;                     // access with unnormalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeBorder; // wrap texture coordinates
	tex.addressMode[1] = hipAddressModeBorder;
	tex.addressMode[2] = hipAddressModeBorder;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex, d_data, desc));

	checkCudaErrors(hipMalloc((void **)&d_transformed, plane_size));
	checkCudaErrors(hipMalloc((void **)&d_inverse, 12 * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_inverse, h_inverse, 12 * sizeof(float), hipMemcpyHostToDevice));

	int nStreams = 2;
	hipStream_t *streams = (hipStream_t *)malloc(nStreams * sizeof(hipStream_t));
	int streamIdx;
	for(streamIdx = 0; streamIdx < nStreams; streamIdx++)
		hipStreamCreate(&streams[streamIdx]);

	// transform the mask
	if(createTransformedMasks) {
		streamIdx = 0;
		hipStream_t stream = streams[streamIdx];
		dim3 threads(32, 32);
		dim3 grid(iDivUp(tw, threads.x), iDivUp(th, threads.y));
		transform_mask_kernel<<<grid, threads, 0, stream>>>(d_transformed, td / 2, w, h, d, zspacing, tw, th, border, d_inverse);
		getLastCudaError("transform_mask_kernel<<<>>> execution failed\n");
		checkCudaErrors(hipMemcpyAsync(h_transformed, d_transformed, plane_size, hipMemcpyDeviceToHost, stream));
		checkCudaErrors(hipStreamSynchronize(stream));
		FILE *maskout = fopen(maskfile, "wb");
		fwrite(h_transformed, sizeof(unsigned short), tw * th, maskout);
		fclose(maskout);
	}

	// transform the data
	FILE *out = fopen(outfile, "wb");
	long start = GetTickCount();
	for(int z = 0; z < td; z++) {
		streamIdx = z % nStreams;
		hipStream_t stream = streams[streamIdx];

		// save the data before overwriting
		if(z >= nStreams) {
			checkCudaErrors(hipStreamSynchronize(stream));
			fwrite(h_transformed, sizeof(unsigned short), tw * th, out);
		}

		// launch the kernel
		dim3 threads(32, 32);
		dim3 grid(iDivUp(tw, threads.x), iDivUp(th, threads.y));
		transform_data_kernel<<<grid, threads, 0, stream>>>(d_transformed, z, w, h, d, tw, th, d_inverse);
		getLastCudaError("transform_data_kernel<<<>>> execution failed\n");
		checkCudaErrors(hipMemcpyAsync(h_transformed, d_transformed, plane_size, hipMemcpyDeviceToHost, stream));
	}
	streamIdx = (streamIdx + 1) % nStreams;
	hipStream_t stream = streams[streamIdx];

	// save the data before overwriting
	checkCudaErrors(hipStreamSynchronize(stream));
	fwrite(h_transformed, sizeof(unsigned short), tw * th, out);


	long end = GetTickCount();
	printf("needed %d ms\n", (end - start));
	fclose(out);

	hipUnbindTexture(tex);
	hipFreeArray(d_data);
	hipFree(d_transformed);
	hipFree(d_inverse);
	hipHostFree(h_transformed);
	for(streamIdx = 0; streamIdx < nStreams; streamIdx++)
		checkCudaErrors(hipStreamDestroy(streams[streamIdx]));
}

void invert3x3(float *mat)
{
	double sub00 = mat[5] * mat[10] - mat[6] * mat[9];
	double sub01 = mat[4] * mat[10] - mat[6] * mat[8];
	double sub02 = mat[4] * mat[9] - mat[5] * mat[8];
	double sub10 = mat[1] * mat[10] - mat[2] * mat[9];
	double sub11 = mat[0] * mat[10] - mat[2] * mat[8];
	double sub12 = mat[0] * mat[9] - mat[1] * mat[8];
	double sub20 = mat[1] * mat[6] - mat[2] * mat[5];
	double sub21 = mat[0] * mat[6] - mat[2] * mat[4];
	double sub22 = mat[0] * mat[5] - mat[1] * mat[4];
	double det = mat[0] * sub00 - mat[1] * sub01 + mat[2] * sub02;
	
	mat[0]  =  (float)(sub00 / det);
	mat[1]  = -(float)(sub10 / det);
	mat[2]  =  (float)(sub20 / det);
	mat[4]  = -(float)(sub01 / det);
	mat[5]  =  (float)(sub11 / det);
	mat[6]  = -(float)(sub21 / det);
	mat[8]  =  (float)(sub02 / det);
	mat[9]  = -(float)(sub12 / det);
	mat[10] =  (float)(sub22 / det);
}

void invert(float *mat)
{
	float dx = mat[3];
	float dy = mat[7];
	float dz = mat[11];
	invert3x3(mat);

	mat[3]  = mat[0] * dx + mat[1] * dy + mat[2]  * dz;
	mat[7]  = mat[4] * dx + mat[5] * dy + mat[6]  * dz;
	mat[11] = mat[8] * dx + mat[9] * dy + mat[10] * dz;
}

static void read_dimensions(const char *dimfile, int *dims)
{
	char buffer[256];
	char *p;
	FILE *f = fopen(dimfile, "r");
	fgets(buffer, 256, f);
	p = strchr(buffer, ':') + 1;
	dims[0] = atoi(p);
	fgets(buffer, 256, f);
	p = strchr(buffer, ':') + 1;
	dims[1] = atoi(p);
	fgets(buffer, 256, f);
	p = strchr(buffer, ':') + 1;
	dims[2] = atoi(p);

	fclose(f);
}

static void read_transformation(const char *regfile, float *matrix)
{
	FILE *f = fopen(regfile, "r");
	char buffer[256];
	char *p;
	int i;
	float zscaling;
	for(i = 0; i < 12; i++) {
		fgets(buffer, 256, f);
		p = strchr(buffer, ':') + 1;
		matrix[i] = atof(p);
	}
	while(fgets(buffer, 256, f) != NULL) {
		if(!strncmp(buffer, "z-scaling", 9)) {
			p = strchr(buffer, ':') + 1;
			zscaling = atof(p);
		}
	}
	fclose(f);
	
	matrix[2]  *= zscaling;
	matrix[6]  *= zscaling;
	matrix[10] *= zscaling;
}

void
apply(const float *mat, float x, float y, float z, float *result)
{
	result[0] = mat[0] * x + mat[1] * y + mat[2]  * z + mat[3];
	result[1] = mat[4] * x + mat[5] * y + mat[6]  * z + mat[7];
	result[2] = mat[8] * x + mat[9] * y + mat[10] * z + mat[11];
}

static void min(float *x, float *min)
{
	if(x[0] < min[0]) min[0] = x[0];
	if(x[1] < min[1]) min[1] = x[1];
	if(x[2] < min[2]) min[2] = x[2];
}

static void max(float *x, float *max)
{
	if(x[0] > max[0]) max[0] = x[0];
	if(x[1] > max[1]) max[1] = x[1];
	if(x[2] > max[2]) max[2] = x[2];
}

void transform_spim(const char *spimdir, const char **filenames, int nFiles)
{
	float mini[3];
	float maxi[3];
	float result[3];
	int i;
	char path[1024];
	float **matrices = (float **)malloc(nFiles * sizeof(float *));
	int **dims = (int **)malloc(nFiles * sizeof(int *));

	for(i = 0; i < nFiles; i++) {
		sprintf(path, "%s/registration/%s%s", spimdir, filenames[i], ".registration");
		printf("path = %s\n", path);
		matrices[i] = (float *)malloc(12 * sizeof(float));
		float *mat = matrices[i];
		read_transformation(path, mat);
		for(int j = 0; j < 3; j++)
			printf("\t%f\t%f\t%f\t%f\n", mat[j * 4], mat[j * 4 + 1], mat[j * 4 + 2], mat[j * 4 + 3]);

		sprintf(path, "%s/registration/%s%s", spimdir, filenames[i], ".dim");

		dims[i] = (int *)malloc(sizeof(int));
		read_dimensions(path, dims[i]);
		printf("\tdims: %d, %d, %d\n", dims[i][0], dims[i][1], dims[i][2]);
		apply(mat, 0, 0, 0, result);
		min(result, mini);
		max(result, maxi);
		apply(mat, dims[i][0], dims[i][1], dims[i][2], result);
		min(result, mini);
		max(result, maxi);
	}

	int tw = (int)(maxi[0] - mini[0] + 0.5);
	int th = (int)(maxi[1] - mini[1] + 0.5);
	int td = (int)(maxi[2] - mini[2] + 0.5);

	for(i = 0; i < nFiles; i++) {
		invert(matrices[i]);
		matrices[i][3]  += mini[0];
		matrices[i][7]  += mini[1];
		matrices[i][11] += mini[2];

		/*
		 * data need be read as raw !?!
		 * sprintf(path, "%s/registered/%s.raw");
		 * transform_cuda(data, dims[i][0], dims[i][1], dims[i][2], tw, th, td, matrices[i], path);
		 */

		free(matrices[i]);
	}

	free(matrices);
}

int test_transform(int argc, char **argv)
{
	printf("%s\n", argv[0]);
	const char *infile = "v0.raw";
	const int w = 1698;
	const int h = 1410;
	const int d = 210;
	const int tw = w;
	const int th = h;
	const int td = d;
	unsigned short *data;
	checkCudaErrors(hipHostMalloc((void**)&data, w * h * d * sizeof(unsigned short)));
	FILE *f = fopen(infile, "rb");
	fread(data, sizeof(unsigned short), w * h * d, f);
	fclose(f);
	float *mat = (float *)malloc(12 * sizeof(float));
	mat[0] = 1; mat[1] = 0; mat[2]  = 0; mat[3]  = 20;
	mat[4] = 0; mat[5] = 1; mat[6]  = 0; mat[7]  = 0;
	mat[8] = 0; mat[9] = 0; mat[10] = 1; mat[11] = 0;
	invert(mat);
	// transform_cuda(data, w, h, d, tw, th, td, mat, "v0.out.raw");

	int dims[3];
	read_dimensions("E:\\SPIM5_Deconvolution\\m6\\cropped\\registration\\v0.tif.dim", dims);
	printf("%d, %d, %d\n", dims[0], dims[1], dims[2]);

	const char *filenames[8] = {"v0.tif", "v1.tif", "v2.tif", "v3.tif", "v4.tif", "v5.tif", "v6.tif", "v7.tif"};
	transform_spim("E:\\SPIM5_Deconvolution\\m6\\cropped", filenames, 8);


	free(mat);
	hipHostFree(data);

	hipDeviceReset();
}

