/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "fmvd_utils.h"
#include "fmvd_deconvolve_common.h"
#include "convolutionFFT2D_common.h"
#include "convolutionFFT2D.cuh"


////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
extern "C" void padKernel(
		float *d_Dst,
		float *d_DstHat,
		float *d_Src,
		int fftH,
		int fftW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	assert(d_Src != d_Dst);
	dim3 threads(32, 8);
	dim3 grid(iDivUp(kernelW, threads.x), iDivUp(kernelH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padKernel_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_DstHat,
			d_Src,
			fftH,
			fftW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padKernel_kernel<<<>>> execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
extern "C" void padDataClampToBorder(
		float *d_estimate,
		data_t *d_Dst,
		data_t *d_Src,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelW,
		int kernelH,
		int nViews,
		hipStream_t stream
		)
{
	assert(d_Src != d_Dst);
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padDataClampToBorder_kernel<<<grid, threads, 0, stream>>>(
			d_estimate,
			d_Dst,
			d_Src,
			fftH,
			fftW,
			dataH,
			dataW,
			kernelH,
			kernelW,
			kernelY,
			kernelX,
			nViews
			);
	getLastCudaError("padDataClampToBorder_kernel<<<>>> execution failed\n");
}

extern "C" void unpadData(
		data_t *d_Dst,
		float *d_Src,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(dataW, threads.x),
			iDivUp(dataH, threads.y));

	unpadData_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_Src,
			fftH,
			fftW,
			dataH,
			dataW
			);
	getLastCudaError("unpadData_kernel<<<>>> execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
extern "C" void modulateAndNormalize(
		fComplex *d_Dst,
		fComplex *d_Src,
		int fftH,
		int fftW,
		int padding,
		hipStream_t stream
		)
{
	assert(fftW % 2 == 0);
	const int dataSize = fftH * (fftW / 2 + padding);

	modulateAndNormalize_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_Dst,
			d_Src,
			dataSize,
			1.0f / (float)(fftW *fftH)
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

extern "C" void divide(
		data_t *d_a,
		float *d_b,
		float *d_dest,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	const int dataSize = fftH * fftW;

	divide_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_a,
			d_b,
			d_dest,
			dataSize
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

extern "C" void mul(
		float *d_a,
		float *d_b,
		float *d_dest,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	const int dataSize = fftH * fftW;

	multiply_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_a,
			d_b,
			d_dest,
			dataSize
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

