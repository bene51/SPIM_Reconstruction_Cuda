/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "fmvd_utils.h"
#include "fmvd_deconvolve_common.h"
#include "convolutionFFT2D_common.h"
#include "convolutionFFT2D.cuh"


////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
extern "C" void padKernel(
		float *d_Dst,
		float *d_Src,
		int fftH,
		int fftW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	assert(d_Src != d_Dst);
	dim3 threads(32, 8);
	dim3 grid(iDivUp(kernelW, threads.x), iDivUp(kernelH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padKernel_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_Src,
			fftH,
			fftW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padKernel_kernel<<<>>> execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
extern "C" void padDataClampToBorderFloat(
		float *d_PaddedData,
		float *d_Data,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	assert(d_PaddedData != d_Data);
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padDataClampToBorderFloat_kernel<<<grid, threads, 0, stream>>>(
			d_PaddedData,
			d_Data,
			fftH,
			fftW,
			dataH,
			dataW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padDataClampToBorder_kernel<<<>>> execution failed\n");
}

extern "C" void padDataClampToBorderAndInitialize(
		float *d_estimate,
		data_t *d_Dst,
		data_t *d_Src,
		float *d_Weights,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelW,
		int kernelH,
		hipStream_t stream
		)
{
	assert(d_Src != d_Dst);
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padDataClampToBorderAndInitialize_kernel<<<grid, threads, 0, stream>>>(
			d_estimate,
			d_Dst,
			d_Src,
			d_Weights,
			fftH,
			fftW,
			dataH,
			dataW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padDataClampToBorderAndInitialize_kernel<<<>>> execution failed\n");
}

extern "C" void padWeights(
		float *d_PaddedWeights,
		float *d_PaddedWeightSums,
		data_t *d_Weights,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		int kernelH,
		int kernelW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	const int kernelY = kernelH / 2;
	const int kernelX = kernelW / 2;

	padWeights_kernel<<<grid, threads, 0, stream>>>(
			d_PaddedWeights,
			d_PaddedWeightSums,
			d_Weights,
			fftH,
			fftW,
			dataH,
			dataW,
			kernelH,
			kernelW,
			kernelY,
			kernelX
			);
	getLastCudaError("padWeights<<<>>> execution failed\n");
}

extern "C" void normalizeWeights(
		float *d_PaddedWeights,
		float *d_PaddedWeightSums,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(fftW, threads.x),
			iDivUp(fftH, threads.y));

	normalizeWeights_kernel<<<grid, threads, 0, stream>>>(
			d_PaddedWeights,
			d_PaddedWeightSums,
			fftH,
			fftW
			);
	getLastCudaError("normalizeWeights_kernel<<<>>> execution failed\n");
}

extern "C" void unpadData(
		data_t *d_Dst,
		float *d_Src,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(dataW, threads.x),
			iDivUp(dataH, threads.y));

	unpadData_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_Src,
			fftH,
			fftW,
			dataH,
			dataW
			);
	getLastCudaError("unpadData_kernel<<<>>> execution failed\n");
}

extern "C" void unpadDataFloat(
		float *d_Dst,
		float *d_Src,
		int fftH,
		int fftW,
		int dataH,
		int dataW,
		hipStream_t stream
		)
{
	dim3 threads(32, 8);
	dim3 grid(
			iDivUp(dataW, threads.x),
			iDivUp(dataH, threads.y));

	unpadDataFloat_kernel<<<grid, threads, 0, stream>>>(
			d_Dst,
			d_Src,
			fftH,
			fftW,
			dataH,
			dataW
			);
	getLastCudaError("unpadData_kernel<<<>>> execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
extern "C" void modulateAndNormalize(
		fComplex *d_Dst,
		fComplex *d_Src,
		int fftH,
		int fftW,
		int padding,
		hipStream_t stream
		)
{
	assert(fftW % 2 == 0);
	const int dataSize = fftH * (fftW / 2 + padding);

	modulateAndNormalize_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_Dst,
			d_Src,
			dataSize,
			1.0f / (float)(fftW *fftH)
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

extern "C" void divide(
		data_t *d_a,
		float *d_b,
		float *d_dest,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	const int dataSize = fftH * fftW;

	divide_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_a,
			d_b,
			d_dest,
			dataSize
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

extern "C" void mul(
		float *d_a,
		float *d_b,
		float *d_weights,
		float *d_dest,
		int fftH,
		int fftW,
		hipStream_t stream
		)
{
	const int dataSize = fftH * fftW;

	multiply_kernel<<<iDivUp(dataSize, 256), 256, 0, stream>>>(
			d_a,
			d_b,
			d_weights,
			d_dest,
			dataSize
			);
	getLastCudaError("modulateAndNormalize() execution failed\n");
}

